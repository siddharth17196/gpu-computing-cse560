#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>

#define T_WIDTH 16
#define LENGTH 100

using namespace std;

__global__ void transpose(float *a, float *b, int width)
{
    float result = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for(int k=0; k<width; k++){
        b[k*width + col] = a[row*width + k];
    }
}
__global__ void matmul_shared(float *a, float *b, int width)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    __shared__ float s_a[T_WIDTH][T_WIDTH];
    int row = ty + by*blockDim.y;
    int col = tx + bx*blockDim.x;


    for(int k = 0; k < width/T_WIDTH; ++k)
    {
        s_a[ty][tx] = a[row*width + (k*T_WIDTH + tx)];
        __syncthreads();
        
        for(int p = 0; p < T_WIDTH; ++p){
            b[p][tx] = s_a[ty][p];
        }
        __syncthreads();
    }
}


int main()
{
    static int A[LENGTH][LENGTH];

    int *h_a, *h_trans;
    int *d_a, *d_trans;

    h_a = (int*)malloc(LENGTH*LENGTH*sizeof(int));
	h_trans = (int*)malloc(LENGTH*LENGTH*sizeof(int));

    int k = 0;
    for(int i=0;i<LENGTH;i++)
    {
        for(int j=0;j<LENGTH;j++)
        {
            h_a[k] = A[i][j];
            k+=1;
        }
    }

    hipMalloc((void**)&d_a, LENGTH*LENGTH*sizeof(int));
	hipMalloc((void**)&d_trans, LENGTH*LENGTH*sizeof(int));

	hipMemcpy(d_a, h_a, LENGTH*LENGTH*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

    dim3 dimBlock1(16, 16, 1);
    dim3 dimGrid1(LENGTH/16, LENGTH/16, 1);
	transpose<<<dimGrid1, dimBlock1>>>(d_a, d_trans, LENGTH);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

    // free the memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout  << "Time taken (normal) in ms: " << milliseconds << std::endl;
    
	hipMalloc((void**)&d_a, LENGTH*LENGTH*sizeof(float));
	hipMalloc((void**)&d_trans, LENGTH*LENGTH*sizeof(float));

	hipMemcpy(d_a, h_a, LENGTH*LENGTH*sizeof(float), hipMemcpyHostToDevice);
	/* hipMemcpy(d_b, h_b, LENGTH*LENGTH*sizeof(float), hipMemcpyHostToDevice); */

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
    dim3 threads(16, 16, 1);
    dim3 blocks(LENGTH/16, LENGTH/16, 1);
	matmul_shared<<<blocks, threads>>>(d_a, d_trans, LENGTH);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float millisecond = 0;
	hipEventElapsedTime(&millisecond, start, stop);
	std::cout  << "Time taken (shared memory) in ms: " <<fixed<<millisecond << std::endl;

    return 0;
}

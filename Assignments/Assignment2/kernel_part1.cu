
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <float.h>


__global__ void sdt_compute(unsigned char *img, int *sz, float *sdt, int sz_edge, int width, float *d_min, int start, int val)
{
  int tx = threadIdx.x + blockDim.x*blockIdx.x;
	extern __shared__ int ep[];
  for(int i=start, j=0;i< val; i++){
    ep[j++] = sz[i];
  }
  // ep[threadIdx.x] = sz[threadIdx.x]; 
  __syncthreads();
  float min_dist, dist2;
  min_dist = d_min[tx];
  float _x, _y;
  float sign;
  float dx, dy;
  int x = tx % width;
  int y = tx / width;
  for(int k=0; k<val-start; k++)
  {
    _x = ep[k] % width;
    _y = ep[k] / width;
    dx = _x - x;
    dy = _y - y;
    dist2 = dx*dx + dy*dy;
    if(dist2 < min_dist) min_dist = dist2;
  }
  d_min[tx] = min_dist;

}

__global__ void final_comp(unsigned char *img, float *d_min, float *sdt)
{
  float sign;
  int tx = threadIdx.x + blockDim.x*blockIdx.x;
  sign  = (img[tx] >= 127)? 1.0f : -1.0f;
  sdt[tx] = sign * sqrtf(d_min[tx]);  
}

extern "C" void run_sampleKernel(unsigned char * bitmap, float *sdt, int width, int height)
{
  //Collect all edge pixels in an array
  int sz = width*height;
  int sz_edge = 0;
  for(int i = 0; i<sz; i++) if(bitmap[i] == 255) sz_edge++;
  int *edge_pixels = new int[sz_edge];
  for(int i = 0, j = 0; i<sz; i++) if(bitmap[i] == 255) edge_pixels[j++] = i;
  std::cout<< "\t"<<sz_edge << " edge pixels in the image of size " << width << " x " << height << "\n"<<std::flush;

  int *d_sz;
  float *temp_min;
  unsigned char *d_img;
  float *d_sdt, *d_min;

  temp_min = new float[height*width];

  for(int i=0;i<height*width;i++){
    temp_min[i] = FLT_MAX;
  }

  hipMalloc((void**)&d_sz, sz_edge*sizeof(int));
  hipMalloc((void**)&d_img, height*width*sizeof(unsigned char));
  hipMalloc((void**)&d_sdt, height*width*sizeof(float));
  hipMalloc((void**)&d_min, height*width*sizeof(float));

  hipMemcpy(d_img, bitmap, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_min, temp_min, width*height*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sz, edge_pixels, sz_edge*sizeof(int), hipMemcpyHostToDevice);

  int divisions = 20;
  int val_div = sz_edge/divisions;
  int n;
  for(n =0; n<divisions; n++){
    sdt_compute<<<(height*width)/256, 256, val_div*sizeof(int)>>>(d_img, d_sz, d_sdt, sz_edge, width, d_min, n*val_div, (n+1)*val_div); 
  }
  if((sz_edge%divisions) !=0){
    sdt_compute<<<(height*width)/256, 256, (sz_edge%divisions)*sizeof(int)>>>(d_img, d_sz, d_sdt, sz_edge, width, d_min, n*val_div, n*val_div + sz_edge%divisions);
  }
  final_comp<<<(height*width)/256, 256>>>(d_img, d_min, d_sdt);
  hipDeviceSynchronize();
  hipMemcpy(sdt, d_sdt, height*width*sizeof(float), hipMemcpyDeviceToHost);

}

